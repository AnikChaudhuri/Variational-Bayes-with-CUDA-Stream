#include "hip/hip_runtime.h"
// This code implements the example shown in the Variational Bayes page on Wikipedia. Link: https://en.wikipedia.org/wiki/Variational_Bayesian_methods
// Here two kernels named nor and gam were launched in parallel with CUDA stream.
// Normal samples were drawn from a normal distribution and stored in vector ex_a.
//These samples were used to estimate the mean and standard deviation by using Variational Bayes algorithm.
//Run this code by using nvcc VB.cu
// The output is the mean and standard deviation calculated in each iteration.

#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

__global__ void nor(double* dlambdan, double lambda0, int dim, int gene, double mu_n, double x_sq, double* dbn, double mu0,
                    double b0,double xb, double* dmean, hiprandState_t *d_states){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t state = d_states[idx]; 
    dbn[idx] = b0 + 0.5*((lambda0+(dim*gene))*(1./dlambdan[idx] + (mu_n*mu_n))-2.0*(lambda0*mu0 + xb)*mu_n+(x_sq)+(lambda0*mu0*mu0));
    double u2 = hiprand_normal(&state); //one normal sample
    dmean[idx] = ((1./dlambdan[idx]) * u2) + mu_n;
    
}

//Marsaglia and Tsang sampler (Gamma sampler)
__global__ void gam(double* dlambdan, double lambda0, double dim, double gene, double an, double* dbn, double mu_n,
                    double* dvar, int count2, hiprandState_t *d_states){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    hiprandState_t state = d_states[idx]; 
    dlambdan[idx] = (lambda0+(dim*gene))*(an/dbn[idx]);

    double d22 = (an) -1.0/3.;
    double c2 = 1.0/sqrt(9*d22);
    do{
        
        
        double u13 = hiprand_uniform(&state); 
        double u23 = hiprand_normal(&state); 
        double v2 = pow((1. + c2*u23), 3);
        int j2 = v2 > 0 && log(u13) < 0.5*pow(u23, 2)+d22 - d22*v2+d22*log(v2);
        count2 = j2;
        dvar[idx] = d22*v2/dbn[idx];//samples
        
        
    }while(count2 == 0);
    
}
__global__ void setup(hiprandState_t *d_states, int j)
{
    int id = threadIdx.x+ blockIdx.x * blockDim.x;
    
    hiprand_init(j, id, 0, &d_states[id]);
}

int main(){
    int gene = 30;
    int dim = 20;
    
    std::vector<double> ex_a(dim*gene);
    std::default_random_engine generator;
    std::normal_distribution<double> distribution(10.0,2.0);

    for(int i = 0; i < dim*gene; i++){ex_a[i] = distribution(generator);} 
    //for(int i = 0; i < dim*gene; i++) std::cout << ex_a[i] <<std::endl;
    
    double a = 0.3; double lambda0 = 1; double mu0 = 2; double b0 = 1.0; 

    thrust::device_vector<double> d1(gene*dim); //device vector
    thrust::device_vector<double> d2(gene*dim); //device vector

    d1 = ex_a; //copying host vector to device vector
    double xb = thrust::reduce(d1.begin(), d1.end()); //adding all the samples
    thrust::transform(d1.begin(), d1.end(), d2.begin(), thrust::square<double>()); //squaring all the samples and storing on device vector d2
    double x_sq = thrust::reduce(d2.begin(), d2.end()); //adding all the elements in device vector d2
    

    double an = a + ((dim*gene)+1)/2.0; double* bn = new double[1]; bn[0] = 0.2;

    double xbar = xb/(dim*gene);
    double mu_n = (lambda0*mu0 + dim*gene*xbar)/(lambda0 + (dim*gene));
    
    double* lambdan = new double[1]; lambdan[0] = 0.02;
    double* dlambdan; 
    hipMalloc(&dlambdan, sizeof(double)*1);//device vector memory allocation
    double* dbn; 
    hipMalloc(&dbn, sizeof(double)*1);  //device vector memory allocation
    double* dmean;
    hipMalloc(&dmean, sizeof(double)*1); //device vector memory allocation
    double* meancopy = new double[1];

    double* varcopy = new double[1];
    double* dvar = new double[1];
    hipMalloc(&dvar, sizeof(double)*1);

    hipMemcpy(dbn, bn, sizeof(double)*1, hipMemcpyHostToDevice); //copying from host to device
    hipMemcpy(dlambdan, lambdan, sizeof(double)*1, hipMemcpyHostToDevice); //copying from host to device
    
    //Random number
    hiprandState_t* d_states;
    hipMalloc(&d_states, sizeof(hiprandState_t)*1);
  
    //Streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1); hipStreamCreate(&stream2); 
    int count2 = 0; 
    
    for(int i =0; i < 1000; i++){
        setup<<<1,1>>>(d_states,i);//setting up random numbers

        nor<<<1,1,0,stream1>>>(dlambdan, lambda0, dim, gene, mu_n, x_sq, dbn, mu0,b0,xb, dmean, d_states); //Normal distribution sampler in stream1

        gam<<<1,1,0,stream2>>>(dlambdan, lambda0, dim, gene, an, dbn, mu_n, dvar, count2, d_states); //Gamma distribution sampler in stream2

        hipMemcpy(bn, dbn, sizeof(double)*1, hipMemcpyDeviceToHost); //copying from device to host
        //hipMemcpy(lambdan, dlambdan, sizeof(double)*1, hipMemcpyDeviceToHost);
        hipMemcpy(meancopy, dmean, sizeof(double)*1, hipMemcpyDeviceToHost); //copying from device to host
        hipMemcpy(varcopy, dvar, sizeof(double)*1, hipMemcpyDeviceToHost); //copying from device to host

        
        std::cout<<"mean is "<< meancopy[0] <<" and s.d. is " << sqrt(1/varcopy[0]) << std::endl;

    }
    
    hipFree(dbn);
    hipFree(lambdan);
    hipFree(dmean);
    hipFree(dvar);
}
